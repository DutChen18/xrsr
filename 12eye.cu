#include "hip/hip_runtime.h"
#include "xrsr.h"
#include <stdint.h>
#include <stdio.h>
#include <inttypes.h>
#include <time.h>

#ifndef GRID_SIZE
#define GRID_SIZE ((uint64_t) 1 << 32)
#endif

__constant__ XRSRMAT skip760;
__managed__ uint64_t results[256];
__managed__ uint64_t count;

__global__ void filter(uint64_t start, uint64_t size)
{
	uint64_t index = (uint64_t) blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t stride = (uint64_t) blockDim.x * gridDim.x;
	uint64_t i = index;
	while (i < size) {
		XRSR128 rng;
		xrsr_seed(&rng, start + i);
		xrsr128_comb(&rng, &skip760);
		for (int j = 0; j < 12; j++)
			if (xrsr_long(&rng) >= 1844674847175606272ULL)
				goto end;
		results[atomicAdd((unsigned long long *) &count, 1)] = start + i;
	end:
		i += stride;
	}
}

void init(void)
{
	XRSRMAT mat;
	XRSR128 tmp;

	xrsr_init();
	xrsr128_init(&tmp, 760, 0);
	xrsrmat_init(&mat);
	xrsrmat_skip(&mat, &tmp);

	hipMemcpyToSymbol(HIP_SYMBOL(skip760), &mat, sizeof(mat), 0, hipMemcpyHostToDevice);
}

void run(uint64_t start, uint64_t size)
{
	uint64_t blockSize = 256;
	uint64_t numBlocks = (size + blockSize - 1) / blockSize;

	count = 0;
	filter<<<numBlocks, blockSize>>>(start, size);
	hipDeviceSynchronize();

	for (uint64_t i = 0; i < count; i++)
		printf("%" PRIu64 "\n", results[i]);
}

int main(int argc, char **argv)
{
	init();
	uint64_t i = 0;
	clock_t start = clock();
	do {
		run(i, GRID_SIZE);
		i += GRID_SIZE;
		clock_t end = clock();
		double delta = (double) (end - start) / CLOCKS_PER_SEC;
		double per_sec = GRID_SIZE / delta;
		double time_left = -i / per_sec;
		printf("%.2fB/s %.2fh\n", per_sec / 1000000000.0, time_left / 3600.0);
		start = end;
	} while (i != 0);
	return 0;
}

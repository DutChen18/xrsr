#include "hip/hip_runtime.h"
#include "xrsr.h"
#include "skip.cu"
#include <stdint.h>
#include <stdio.h>
#include <inttypes.h>
#include <time.h>

#ifndef GRID_SIZE
#define GRID_SIZE ((uint64_t) 1 << 32)
#endif

__managed__ uint64_t results[256];
__managed__ uint64_t count;

__global__ void filter(uint64_t start, uint64_t size)
{
	uint64_t index = (uint64_t) blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t stride = (uint64_t) blockDim.x * gridDim.x;
	uint64_t i = index;
	while (i < size) {
		XRSR128 rng;
		xrsr_seed(&rng, start + i);
		skip(&rng);
		for (int j = 0; j < 12; j++)
			if (xrsr_long(&rng) < 16602070326045573120ULL)
				goto end;
		results[atomicAdd((unsigned long long *) &count, 1)] = start + i;
	end:
		i += stride;
	}
}

void run(uint64_t start, uint64_t size)
{
	uint64_t blockSize = 256;
	uint64_t numBlocks = (size + blockSize - 1) / blockSize / 16;

	count = 0;
	filter<<<numBlocks, blockSize>>>(start, size);
	hipDeviceSynchronize();

	for (uint64_t i = 0; i < count; i++)
		printf("%" PRIu64 "\n", results[i]);
}

int main(int argc, char **argv)
{
	uint64_t i = 0;
	clock_t start = clock();
	do {
		run(i, GRID_SIZE);
		i += GRID_SIZE;
		clock_t end = clock();
		double delta = (double) (end - start) / CLOCKS_PER_SEC;
		double per_sec = GRID_SIZE / delta;
		double time_left = -i / per_sec;
		printf("%.2fB/s %.2fh\n", per_sec / 1000000000.0, time_left / 3600.0);
		start = end;
	} while (i != 0);
	return 0;
}
